#include "hip/hip_runtime.h"
#include <Multiplication.h>

#include <cassert>
#include <utility>
#include "../include/Matrix.h"

#include <iostream>

__global__
static void kernel_multiply_matrix(MatrixRef a, MatrixRef b, MatrixRef c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int line = index / a.order();
    int column = index % a.order();

    for(int k = 0; k < a.order(); k++)
        c[line][column] += a[line][k] * b[k][column];

    return ;
}

Matrix multiply_matrix(Matrix &a, Matrix &b)
{
    assert(a.order() == b.order());
    Matrix c(a.order());

    MatrixRef refC = c.ref();
    kernel_multiply_matrix
        <<<(a.order() * a.order() + 256 - 1) / 256, 256>>>
        (
            a.ref(),
            b.ref(),
            refC
        );
    hipDeviceSynchronize();

    return std::move(c);
}