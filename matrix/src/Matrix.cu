//
// Created by alex on 12/22/18.
//

#include <Matrix.h>

Matrix::Matrix(int n): m_n(n)
{
    hipMallocManaged(&m_matrix, n * n * sizeof(float));
}

SubscriptProxy Matrix::operator[](int line)
{
    return SubscriptProxy(m_n, line, m_matrix);
}

SubscriptProxy::SubscriptProxy(int n, int line, float *matrix):
    m_n(n),
    m_line(line),
    m_matrix(matrix)
{}

SubscriptProxy::SubscriptProxy(SubscriptProxy &&other):
    m_matrix(other.m_matrix),
    m_n(other.m_n),
    m_line(other.m_line)
{
    other.m_matrix = nullptr;
    other.m_line = 0;
    other.m_n = 0;
}

float& SubscriptProxy::operator[](int column)
{
    return m_matrix[m_line * m_n + column];
}