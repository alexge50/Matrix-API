//
// Created by alex on 12/22/18.
//

#include <Matrix.h>
#include <cuda-indexing.h>
#include "../include/Matrix.h"

#include <cstring>
#include <utility>

Matrix::Matrix(int n): m_n(n)
{
    hipMallocManaged(&m_matrix, m_n * m_n * sizeof(float));

    std::memset(m_matrix, 0, m_n * m_n * sizeof(float));
}

Matrix::Matrix(const Matrix &other)
{
    m_n = other.m_n;
    hipMallocManaged(&m_matrix, m_n * m_n * sizeof(float));

    std::memcpy(m_matrix, other.m_matrix, m_n * m_n * sizeof(float));
}

Matrix::Matrix(Matrix &&other):
    m_matrix(other.m_matrix),
    m_n(other.m_n)
{
    other.m_matrix = nullptr;
    other.m_n = 0;
}

Matrix::~Matrix()
{
    hipFree(m_matrix);
}

__host__ __device__ SubscriptProxy Matrix::operator[](int line)
{
    return SubscriptProxy(m_n, line, m_matrix);
}

int Matrix::order() const
{
    return m_n;
}

MatrixRef &&Matrix::ref() {
    return std::move(MatrixRef(m_matrix, m_n));
}

MatrixRef::MatrixRef(float *matrix, int n):
    m_matrix(matrix),
    m_n(n)
{}

__host__ __device__ SubscriptProxy MatrixRef::operator[](int line)
{
    return SubscriptProxy(m_n, line, m_matrix);
}

int MatrixRef::order() const
{
    return m_n;
}

__host__ __device__ SubscriptProxy::SubscriptProxy(int n, int line, float *matrix):
    m_n(n),
    m_line(line),
    m_matrix(matrix)
{}

__host__ __device__ SubscriptProxy::SubscriptProxy(SubscriptProxy &&other):
    m_matrix(other.m_matrix),
    m_n(other.m_n),
    m_line(other.m_line)
{
    other.m_matrix = nullptr;
    other.m_line = 0;
    other.m_n = 0;
}

__host__ __device__ float& SubscriptProxy::operator[](int column)
{
    return m_matrix[m_line * m_n + column];
}