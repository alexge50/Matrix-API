#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/cuda-indexing.h"
#include <Matrix.h>

__global__
void multiply_matrix(int n, Matrix a, Matrix b, Matrix *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int line = index / n;
    int column = index % n;

    for(int k = 0; k < n; k++)
        (*c)[line][column] += a[line][k] * b[k][column];

    return ;
}

int main()
{
    constexpr int n = 100;
    Matrix a(n), b(n), *c;
    hipMallocManaged(&c, sizeof(Matrix));

    new(c)Matrix(n);

    for(int i = 0; i < n; i++)
        for(int j = 0; j < n; j++)
            a[i][j] = b[i][j] = (i == j);

    multiply_matrix<<<(n * n + 256 - 1) / 256, 256>>>(n, a, b, c);
    hipDeviceSynchronize();

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
            std::cout << (*c)[i][j] << " ";
        std::cout << "\n";
    }

    hipFree(c);
    return 0;
}