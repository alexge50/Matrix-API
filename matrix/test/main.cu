#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/cuda-indexing.h"
#include <Matrix.h>

/*
__global__
void multiply_matrix(int n, float *a, float *b, float *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int line = index / n;
    int column = index % n;

    for(int k = 0; k < n; k++)
        c[line * n + column] += a[line * n + k] * b[k * n + column];

    return ;
}*/

__global__
void multiply_matrix(int n, Matrix a, Matrix b, Matrix *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int line = index / n;
    int column = index % n;

    for(int k = 0; k < n; k++)
    (*c)[line][column] += a[line][k] * b[k][column];

    return ;
}

int main()
{
    int n = 100;
    Matrix a(100), b(100), *c;
    hipMallocManaged(&c, sizeof(Matrix));

    new(c)Matrix(100);

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            a[i][j] = (i + j) * 0.01f;
            b[i][j] = (i * i + j * j + 0.2f) * 0.001f;
        }
    }

    multiply_matrix<<<(n + 128 - 1) / 128, 128>>>(n, a, b, c);
    hipDeviceSynchronize();

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
            std::cout << (*c)[i][j] << " ";
        std::cout << "\n";
    }

    hipFree(c);
    return 0;
}