#include "hip/hip_runtime.h"
#include <iostream>
#include "../src/cuda-indexing.h"
#include <Matrix.h>

__global__
void multiply_matrix(int n, float *a, float *b, float *c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int line = index / n;
    int column = index % n;

    for(int k = 0; k < n; k++)
        c[line * n + column] += a[line * n + k] * b[k * n + column];

    return ;
}

int main()
{
    int n = 10000;
    float *a, *b, *c;
    hipMallocManaged(&a, n * n * sizeof(float));
    hipMallocManaged(&b, n * n * sizeof(float));
    hipMallocManaged(&c, n * n * sizeof(float));

    for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
        {
            a[i * n + j] = (i + j) * 0.01f;
            b[i * n + j] = (i * i + j * j + 0.2f) * 0.001f;
        }
    }

    multiply_matrix<<<(n + 128 - 1) / 128, 128>>>(n, a, b, c);
    hipDeviceSynchronize();

    /*for(int i = 0; i < n; i++)
    {
        for(int j = 0; j < n; j++)
            std::cout << b[i * n + j] << " ";
        std::cout << "\n";
    }*/

    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
}